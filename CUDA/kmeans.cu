#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes CUDA and helper functions
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "template.h"

#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/pair.h>
#include <thrust/sequence.h>
#include <thrust/fill.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/tuple.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
//#include <unistd.h>

#define NUMBER_OF_POINTS 500
#define NUMBER_OF_CENTROIDS 10
#define THREADS_PER_BLOCK 128
#define THREAD_PER_DIM 16
#define LOOP_COUNT 2

#define SEED 125
#define MIN_FLOAT 0
#define MAX_FLOAT 20

typedef struct Point {
	float X;
	float Y;
	float Z;
	int inClaster;
} Point;

int CountSumIterations(int a)
{
	a -= 1;
	int max2 = 0;
	for (int i = 32 - 1; i >= 0; i--)
	{
		int bit = (a >> i) & 1;
		if (bit == 1)
		{
			max2 = i;
			break;
		}
	}
	return max2 + 1;
}

void FillCoordinatesTab(Point* points, Point* centroids, int n, int k, float* hostPointsX, float* hostPointsY, float* hostPointsZ, int* hostPointsSum)
{
	for (int i = 0; i < n; i++)
	{
		float x = MIN_FLOAT + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX/(MAX_FLOAT - MIN_FLOAT)));
		float y  = MIN_FLOAT + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX/(MAX_FLOAT - MIN_FLOAT)));
		float z = MIN_FLOAT + static_cast<float>(rand()) / (static_cast<float>(RAND_MAX/(MAX_FLOAT - MIN_FLOAT)));

		points[i].X = x; hostPointsX[i] = x;
		points[i].Y = y; hostPointsY[i] = y;
		points[i].Z = z; hostPointsZ[i] = z;
		points[i].inClaster = -1;
		hostPointsSum[i] = 1;
	}

	//for (int i = 0; i < n; i++) {
	//	Point p = points[i];
	//	printf("(%f,%f,%f)\n", p.X, p.Y, p.Z);
	//}
	for (int i = 0; i < k; i++) {
		centroids[i] = points[i];
	}
}
void InitalizeCentroids(Point* hostInitialCentroids, Point* hostCentroids, int k)
{
	for (int i = 0; i < k; i++) {
		hostCentroids[i] = hostInitialCentroids[i];
	}
}

void PrintClusters(int n, int k, Point* hostClusters)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < k; j++)
		{
			Point p = hostClusters[k * i + j];
			printf("(%f,%f,%f,%d) | ", p.X, p.Y, p.Z, p.inClaster);
		}
		printf("\n");

	}

}

__host__ __device__ inline static
float Distance(Point point, Point centroid)
{
	float dX = point.X - centroid.X;
	float dY = point.Y - centroid.Y;
	float dZ = point.Z - centroid.Z;
	return dX * dX + dY * dY + dZ * dZ;
}
__host__ __device__ inline static
float Distance(float x, float y, float z, Point centroid)
{
	float dX = x - centroid.X;
	float dY = y - centroid.Y;
	float dZ = z - centroid.Z;
	return dX * dX + dY * dY + dZ * dZ;
}

__global__ void
ClearClustersTab(int clustersMemory, Point* clusters)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= clustersMemory) return;

	clusters[threadId].X = 0;
	clusters[threadId].Y = 0;
	clusters[threadId].Z = 0;
	clusters[threadId].inClaster = 0;
}

__global__ void
SumClusters (int k, int iterationNumber, int step, Point* deviceClusters, int n)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= step) return;
	int nk = n * k;

	for (int i = 0; i < iterationNumber; i++)
	{
		for (int j = 0; j < k; j++)
		{
			int index1 = threadId * k + j;
			int index2 = (threadId + step) * k + j;
			if (index2 < nk)
			{
				deviceClusters[index1].X += deviceClusters[index2].X;
				deviceClusters[index1].Y += deviceClusters[index2].Y;
				deviceClusters[index1].Z += deviceClusters[index2].Z;
				deviceClusters[index1].inClaster += deviceClusters[index2].inClaster;
				//printf("ThreadId %d,index(%d , %d) InClaster %d\n", threadId, index1, index2, deviceClusters[index2].inClaster);
			}
		}

		__syncthreads();
		step /= 2;
		if (threadId >= step) return;
	}
}

__global__ void
FindNearestCentroidsScatter(int n, int numOfCentroids, Point* points, Point* centroids, Point* clusters)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= n) return;

	Point point = points[threadId];

	//Finding nearest centroid for given point
	float minDist = Distance(point, centroids[0]);
	int nearestCentroidIndex = 0;

	for (int i=1; i < numOfCentroids; i++)
	{

		float dist = Distance(point, centroids[i]);
		if (dist < minDist) 
		{
			minDist = dist;
			nearestCentroidIndex = i;
		}
	}
	point.inClaster = 1;

	clusters[threadId * numOfCentroids + nearestCentroidIndex] = point;
	//printf("Thread[%d, %d = %d], (%f,%f,%f, %d), {%d}\n",
	//	blockIdx.x, threadIdx.x, threadId, point.X, point.Y, point.Z, point.inClaster, nearestCentroidIndex);
}


__global__ void
FindNearestCentroidsReduce(int n, int numOfCentroids, Point* centroids, float* pointsX, float* pointsY, float* pointsZ, int* keys)
{
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= n) return;

	float x = pointsX[threadId]; float y = pointsY[threadId]; float z = pointsZ[threadId];

	//Finding nearest centroid for given point
	float minDist = Distance(x, y, z, centroids[0]);
	int nearestCentroidIndex = 0;

	for (int i=1; i < numOfCentroids; i++)
	{
		float dist = Distance(x, y, z, centroids[i]);
		if (dist < minDist) 
		{
			minDist = dist;
			nearestCentroidIndex = i;
		}
	}

	keys[threadId] = nearestCentroidIndex;
	//printf("Thread[%d, %d = %d], (%f,%f,%f, %d), {%d}\n",
	//	blockIdx.x, threadIdx.x, threadId, x, y, z, 1, nearestCentroidIndex);
}


__device__ void warpReduce(volatile float* sdata, int tid) {
sdata[tid * 4] += sdata[(tid + 16) * 4];
sdata[tid * 4 + 1] += sdata[(tid + 16) * 4 + 1];
sdata[tid * 4 + 2] += sdata[(tid + 16) * 4 + 2];
sdata[tid * 4 + 3] += sdata[(tid + 16) * 4 + 3];

sdata[tid * 4] += sdata[(tid + 8) * 4];
sdata[tid * 4 + 1] += sdata[(tid + 8) * 4 + 1];
sdata[tid * 4 + 2] += sdata[(tid + 8) * 4 + 2];
sdata[tid * 4 + 3] += sdata[(tid + 8) * 4 + 3];

sdata[tid * 4] += sdata[(tid + 4) * 4];
sdata[tid * 4 + 1] += sdata[(tid + 4) * 4 + 1];
sdata[tid * 4 + 2] += sdata[(tid + 4) * 4 + 2];
sdata[tid * 4 + 3] += sdata[(tid + 4) * 4 + 3];

sdata[tid * 4] += sdata[(tid + 2) * 4];
sdata[tid * 4 + 1] += sdata[(tid + 2) * 4 + 1];
sdata[tid * 4 + 2] += sdata[(tid + 2) * 4 + 2];
sdata[tid * 4 + 3] += sdata[(tid + 2) * 4 + 3];

sdata[tid * 4] += sdata[(tid + 1) * 4];
sdata[tid * 4 + 1] += sdata[(tid + 1) * 4 + 1];
sdata[tid * 4 + 2] += sdata[(tid + 1) * 4 + 2];
sdata[tid * 4 + 3] += sdata[(tid + 1) * 4 + 3];
}

__global__ void
FindNearestCentroidsGather(int n, int k, Point* points, Point* centroids, int tpb)
{
	extern __shared__ float blockPoints[];
	int blockId = blockIdx.x;
	int threadId = threadIdx.x;
	if (threadId >= n) return;

	for (int pIndex = threadId; pIndex < n; pIndex += tpb)
	{
		Point p = points[pIndex];

		float minDist = Distance(p, centroids[0]);
		int nearestCentroidIndex = 0;
		for (int i=1; i < k; i++)
		{
			float dist = Distance(p, centroids[i]);
			if (dist < minDist) 
			{
				minDist = dist;
				nearestCentroidIndex = i;
			}
		}

		if (nearestCentroidIndex == blockId)
		{
			blockPoints[threadId * 4] += p.X;
			blockPoints[threadId * 4 + 1] += p.Y;
			blockPoints[threadId * 4 + 2] += p.Z;
			blockPoints[threadId * 4 + 3] += 1;
		}
	}
	__syncthreads();
	//printf("Block %d, Thread %d CentroidInfo(%f,%f,%f,%f) \n", blockId, threadId,
	//	blockPoints[threadId * 4],blockPoints[threadId * 4 + 1],blockPoints[threadId * 4 + 2],blockPoints[threadId * 4 + 3]);
	int step = tpb / 2;
	do {
		if (threadId >= step) return;
		int ind1 = threadId * 4;
		int ind2 = (threadId + step) * 4;
		blockPoints[ind1] += blockPoints[ind2];
		blockPoints[ind1 + 1] += blockPoints[ind2 + 1];
		blockPoints[ind1 + 2] += blockPoints[ind2 + 2];
		blockPoints[ind1 + 3] += blockPoints[ind2 + 3];
		step /= 2;
	} while (step >= 1);

	centroids[blockId].X = blockPoints[0] / blockPoints[3];
	centroids[blockId].Y = blockPoints[1] / blockPoints[3];
	centroids[blockId].Z = blockPoints[2] / blockPoints[3];
	centroids[blockId].inClaster = blockPoints[3];
}


typedef thrust::tuple<float,float,float, int> Float3;
struct SumTuples : public thrust::binary_function<Float3,Float3,Float3>
{
    __host__ __device__
        Float3 operator()(const Float3& a, const Float3& b) const
        {
			Float3 temp;
			thrust::get<0>(temp) = thrust::get<0>(a)+thrust::get<0>(b);
			thrust::get<1>(temp) = thrust::get<1>(a)+thrust::get<1>(b);
			thrust::get<2>(temp) = thrust::get<2>(a)+thrust::get<2>(b);
			thrust::get<3>(temp) = thrust::get<3>(a)+thrust::get<3>(b);
			return temp;
        }
};

int main(int argc, char **argv)
{

    StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);

    srand(SEED);
    int n = NUMBER_OF_POINTS;
	int k = NUMBER_OF_CENTROIDS;
	int tpb = THREADS_PER_BLOCK;

	//Coordinates tab memory allocation and filling
    int pointsMemory = sizeof(Point) * n;
	int centroidsMemory = sizeof(Point) * NUMBER_OF_CENTROIDS;
	int clustersMemory = pointsMemory * NUMBER_OF_CENTROIDS;
    Point* hostPoints = (Point*)malloc(pointsMemory);
	int xPointsMemory = sizeof(float) * n;
	int keysMemory = sizeof(int) * n;
    float* hostPointsX = (float*)malloc(xPointsMemory);
    float* hostPointsY = (float*)malloc(xPointsMemory);
    float* hostPointsZ = (float*)malloc(xPointsMemory);
    int* hostPointsSum = (int*)malloc(keysMemory);
    int* hostKeys = (int*)malloc(keysMemory);

	Point* hostCentroids = (Point*)malloc(centroidsMemory);
	Point* hostResultCentroids = (Point*)malloc(centroidsMemory);
	Point* hostInitialCentroids = (Point*)malloc(centroidsMemory);
    Point* hostClusters = (Point*)malloc(clustersMemory);
	printf("Allocated memory in bytes: %d\n", pointsMemory + centroidsMemory + clustersMemory);

	FillCoordinatesTab(hostPoints, hostInitialCentroids, n, k, hostPointsX, hostPointsY, hostPointsZ, hostPointsSum);
	InitalizeCentroids(hostInitialCentroids, hostCentroids, k);

	// Copy memory to GPU
    Point* devicePoints; Point* deviceCentroids; Point* deviceClusters;
    checkCudaErrors(hipMalloc((void **) &devicePoints, pointsMemory));
    checkCudaErrors(hipMemcpy(devicePoints, hostPoints, pointsMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &deviceCentroids, centroidsMemory));
    checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &deviceClusters, clustersMemory));
    checkCudaErrors(hipMemcpy(deviceClusters, hostClusters, clustersMemory, hipMemcpyHostToDevice));
    if(!hostPoints || !devicePoints || !hostClusters || !deviceClusters || !hostCentroids || !deviceCentroids)
		printf("Memory allocation failed\n");

	float* devicePointsX; float* devicePointsY; float* devicePointsZ; int* deviceKeys; int* devicePointsSum;
    checkCudaErrors(hipMalloc((void **) &devicePointsX, xPointsMemory));
    checkCudaErrors(hipMemcpy(devicePointsX, hostPointsX, xPointsMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devicePointsY, xPointsMemory));
    checkCudaErrors(hipMemcpy(devicePointsY, hostPointsY, xPointsMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devicePointsZ, xPointsMemory));
    checkCudaErrors(hipMemcpy(devicePointsZ, hostPointsZ, xPointsMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &deviceKeys, keysMemory));
    checkCudaErrors(hipMemcpy(deviceKeys, hostKeys, keysMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devicePointsSum, keysMemory));
    checkCudaErrors(hipMemcpy(devicePointsSum, hostPointsSum, keysMemory, hipMemcpyHostToDevice));
    if(!hostPointsX || !devicePointsX || !hostPointsY || !devicePointsY || !hostPointsZ || !devicePointsZ || !hostKeys || !deviceKeys || !hostPointsSum || !devicePointsSum)
		printf("Memory allocation failed 2\n");



	////GATHER METHOD
	printf("\nGATHER METHOD FOR %d LOOPS\n", LOOP_COUNT);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	int blockMemorySize = tpb * 4 * sizeof(float) * 2;
	int loop = 0;
	do 
	{
		FindNearestCentroidsGather <<<k, tpb, blockMemorySize>>> (n, k, devicePoints, deviceCentroids, tpb);
		hipDeviceSynchronize(); getLastCudaError("FindNearestCentroidsGather");

		checkCudaErrors(hipMemcpy(hostCentroids, deviceCentroids, centroidsMemory, hipMemcpyDeviceToHost));
		
		printf("Loop number %d\n", loop);
		for (int i = 0; i < k; i++) 
		{
			Point c = hostCentroids[i];
			std::cout<<"Centroid ["<<i<<"] = (" << c.X << " , " << c.Y << " , " << c.Z << " , " << c.inClaster << ")\n";
		}
		//checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));

	} while (++loop < LOOP_COUNT);
	sdkStopTimer(&timer);
	printf("\nGATHER METHOD processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));



	//SCATTER METHOD
	printf("\nSCATTER METHOD FOR %d LOOPS\n", LOOP_COUNT);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	InitalizeCentroids(hostInitialCentroids, hostCentroids, k);
	checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));
	loop = 0;
	do 
	{
		int numberOfClusterPoints = NUMBER_OF_CENTROIDS * n;
		int numOfBlocks = numberOfClusterPoints / tpb + 1;
		ClearClustersTab <<<numOfBlocks, tpb >>> (numberOfClusterPoints, deviceClusters);
		hipDeviceSynchronize(); getLastCudaError("ClearClustersTab");

		int numberOfBlocks = n / tpb + 1;
		FindNearestCentroidsScatter <<<numberOfBlocks, tpb>>> (n, k, devicePoints, deviceCentroids, deviceClusters);
		hipDeviceSynchronize(); getLastCudaError("FindNearestCentroids");

		//See what got what
		//checkCudaErrors(hipMemcpy(hostClusters, deviceClusters, clustersMemory, hipMemcpyDeviceToHost));
		//PrintClusters(n, k, hostClusters);

		///Summing everything up
		int iterationNumber = CountSumIterations(n);
		int step = (int)pow(2, iterationNumber - 1);
		numberOfBlocks = step / tpb + 1;
		//std::cout << "N is " << n << " and count sum iterations is " << iterationNumber << std::endl;
		//std::cout << "Step" << step << "numofblocks" << numberOfBlocks << "\n";

		SumClusters <<<numberOfBlocks, tpb>>> (k, iterationNumber, step, deviceClusters, n);
		hipDeviceSynchronize(); getLastCudaError("SumClusters");

		checkCudaErrors(hipMemcpy(hostClusters, deviceClusters, clustersMemory, hipMemcpyDeviceToHost));
		//PrintClusters(n, k, hostClusters);

		//Calculating new centroids
		printf("Loop number %d\n", loop);
		for (int i = 0; i < k; i++)
		{
			hostCentroids[i].X = hostClusters[i].X / hostClusters[i].inClaster;
			hostCentroids[i].Y = hostClusters[i].Y / hostClusters[i].inClaster;
			hostCentroids[i].Z = hostClusters[i].Z / hostClusters[i].inClaster;
			Point centroid = hostCentroids[i];
			printf("Centroid [%d] = (%f,%f,%f) of %d points\n", i, centroid.X, centroid.Y, centroid.Z, hostClusters[i].inClaster);
		}
		checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));

	} while (++loop < LOOP_COUNT);
	sdkStopTimer(&timer);
	printf("\nSCATTER METHOD processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));



	//REDUCE_BY_KEY METHOD
	printf("\nREDUCE_BY_KEY METHOD FOR %D LOOPS\n", LOOP_COUNT);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	InitalizeCentroids(hostInitialCentroids, hostCentroids, k);
	checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));

	//Keys
	thrust::device_ptr<int> ptrDeviceKeys = thrust::device_pointer_cast(deviceKeys);
	thrust::device_vector<int> keysResult(k);

	//Points
	thrust::device_ptr<float> ptrDevicePointsX = thrust::device_pointer_cast(devicePointsX);
	thrust::device_ptr<float> ptrDevicePointsY = thrust::device_pointer_cast(devicePointsY);
	thrust::device_ptr<float> ptrDevicePointsZ = thrust::device_pointer_cast(devicePointsZ);
	thrust::device_ptr<int> ptrDevicePointsSum = thrust::device_pointer_cast(devicePointsSum);

	auto ptrDeviceTuples = thrust::make_zip_iterator(thrust::make_tuple(ptrDevicePointsX, ptrDevicePointsY, ptrDevicePointsZ, ptrDevicePointsSum));
	thrust::device_vector<float> valuesResultX(k);
	thrust::device_vector<float> valuesResultY(k);
	thrust::device_vector<float> valuesResultZ(k);
	thrust::device_vector<int> valuesResultSum(k);

	loop = 0;
	do 
	{
		int numberOfBlocks = n / tpb + 1;
		FindNearestCentroidsReduce <<<numberOfBlocks, tpb>>> (n, k, deviceCentroids, devicePointsX, devicePointsY, devicePointsZ, deviceKeys);
		hipDeviceSynchronize(); getLastCudaError("FindNearestCentroidsReduce");

		//Sorting points by centroid indices
		thrust::sort_by_key(ptrDeviceKeys, ptrDeviceKeys + n, ptrDeviceTuples);

		//Reducing points
		thrust::equal_to<int> binary_pred;
		auto new_end = thrust::reduce_by_key(ptrDeviceKeys, ptrDeviceKeys + n,
		ptrDeviceTuples,
		keysResult.begin(),
	    thrust::make_zip_iterator(make_tuple(valuesResultX.begin(), valuesResultY.begin(), valuesResultZ.begin(), valuesResultSum.begin())),
		binary_pred, SumTuples() );
		
		printf("Loop number %d\n", loop);
		for (int i = 0; i < k; i++) 
		{
			int count = valuesResultSum[i];
			float newx = valuesResultX[i] / count;
			float newy = valuesResultY[i] / count;
			float newz = valuesResultZ[i] / count;
			std::cout <<"Centroid ["<< i<<"]=" << ": " << newx << "," << newy << "," << newz <<"of "<<count<<"\n" ;
			hostCentroids[i].X = newx;
			hostCentroids[i].Y = newy;
			hostCentroids[i].Z = newz;
		}
		checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));

	} while (++loop < LOOP_COUNT);
	sdkStopTimer(&timer);
	printf("\nREDUCE_BY_KEY METHOD processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));



	//CPU
	printf("\nCALCULATIONS ON CPU FOR %d LOOPS\n", LOOP_COUNT);
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);

	InitalizeCentroids(hostInitialCentroids, hostCentroids, k);
	loop = 0;
	do 
	{
		for (int i = 0; i < k; i++)
		{
			hostResultCentroids[i].X = 0;
			hostResultCentroids[i].Y = 0;
			hostResultCentroids[i].Z = 0;
			hostResultCentroids[i].inClaster = 0;
		}

		for (int j = 0; j < n; j++)
		{
			float minDist = Distance(hostPoints[j], hostCentroids[0]);
			int nearestCentroidIndex = 0;
			for (int i = 0; i < k; i++)
			{
				float dist = Distance(hostPoints[j], hostCentroids[i]);
				if (dist < minDist) 
				{
					minDist = dist;
					nearestCentroidIndex = i;
				}
			}
			hostResultCentroids[nearestCentroidIndex].X += hostPoints[j].X;
			hostResultCentroids[nearestCentroidIndex].Y += hostPoints[j].Y;
			hostResultCentroids[nearestCentroidIndex].Z += hostPoints[j].Z;
			hostResultCentroids[nearestCentroidIndex].inClaster += 1;
		}
		//Calculating new centroids
		printf("Loop number %d\n", loop);
		for (int i = 0; i < k; i++)
		{
			hostCentroids[i].X = hostResultCentroids[i].X / hostResultCentroids[i].inClaster;
			hostCentroids[i].Y = hostResultCentroids[i].Y / hostResultCentroids[i].inClaster;
			hostCentroids[i].Z = hostResultCentroids[i].Z / hostResultCentroids[i].inClaster;
			Point centroid = hostCentroids[i];
			printf("Centroid [%d] = (%f,%f,%f) of %d points\n", i, centroid.X, centroid.Y, centroid.Z, hostResultCentroids[i].inClaster);
		}
		checkCudaErrors(hipMemcpy(deviceCentroids, hostCentroids, centroidsMemory, hipMemcpyHostToDevice));

	} while (++loop < LOOP_COUNT);
	sdkStopTimer(&timer);
	printf("\nCPU processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));



	//Free memory and timer
    free(hostPoints);
    hipFree(devicePoints);
	sdkDeleteTimer(&timer);
    getLastCudaError("Kernel execution failed");
}
