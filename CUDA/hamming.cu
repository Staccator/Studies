#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes CUDA and helper functions
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
//#include <unistd.h>

// declaration, forward
void runTest(int argc, char **argv);

#define SIZE 3
#define MASK 0x3FF
#define NUMBER_OF_SEQUENCES 20000
#define SEED 137
#define THREAD_PER_BLOCK 256
#define THREAD_PER_DIM 16
#define TestCount 5
//Maximum number of sequences: 2^16

void PrintIntBinary(int a)
{
	for (int i = 32 - 1; i >= 0; i--)
	{
		printf("%d", (a >> i) & 1);
	}
}

int MinimumBiggerPowerOf2(int a)
{
	int max2 = 0;
	for (int i = 32 - 1; i >= 0; i--)
	{
		int bit = (a >> i) & 1;
		if (bit == 1)
		{
			max2 = i;
			break;
		}
	}
	return max2;
}

void FillSequenceTable(int* hostSequenceTab, int n)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < SIZE; j++)
		{
			hostSequenceTab[i * SIZE + j] = rand() & MASK;
		}
	}
}

void FillPairTable(int* hostPairsTab, int n)
{
	int tabIndex = 0;
	unsigned int pair = 0;
	for (int i = 0; i < n - 1; i++)
	{
		for (int j = i + 1; j < n; j++)
		{
			pair = j;
			pair |= ((i & 0x0000FFFF) << 16);
			hostPairsTab[tabIndex++] = pair;
		}
	}
}

int PrintGoodPairsFromArray(int* pairsTab, int* resultTab, int* sequenceTab, int pairsCount)
{
	printf("START OF PRINTING GOOD PAIRS\n");
	int goodPairCount = 0;
	for (int i = 0; i < pairsCount; i++)
	{
		if (resultTab[i] == 1)
		{
			goodPairCount++;
			int pair = pairsTab[i];
			int x = (pair & 0xFFFF0000) >> 16;
			int y = pair & 0x0000FFFF;
			printf("Printing Good Pair where [x is %d and y is %d] \n", x, y);
			for (int i = 0; i < SIZE; i++)
			{
				PrintIntBinary(sequenceTab[x * SIZE + i]);
			}
			printf("\n");
			for (int i = 0; i < SIZE; i++)
			{
				PrintIntBinary(sequenceTab[y * SIZE + i]);
			}
			printf("\n");
		}
	}

	return goodPairCount;
}

void PrintGoodPairsFromStack(int* sequenceTab, unsigned int * stack, unsigned int stackSize)
{
	printf("\nPRINTING GOOD PAIRS FOUND ON GPU\n");
	for (int j = 0; j < stackSize / (TestCount * 3); j += 2)
	{
		int x = stack[j];
		int y = stack[j+1];
		printf("Printing Good Pair where [x is %d and y is %d] \n", x, y);
		for (int i = 0; i < SIZE; i++)
		{
			PrintIntBinary(sequenceTab[x * SIZE + i]);
		}
		printf("\n");
		for (int i = 0; i < SIZE; i++)
		{
			PrintIntBinary(sequenceTab[y * SIZE + i]);
		}
		printf("\n\n");
	}
}


int NumberOfGoodPairs(int* sequenceTab, int n)
{
	int goodPairsCount = 0;
	for (int x = 0; x < n - 1; x++)
	{
		for (int y = x + 1; y < n; y++)
		{
			int count = 0;
			for(int i = 0; i < SIZE; i++)
			{
				unsigned int xorr = sequenceTab[x*SIZE+i] ^ sequenceTab[y*SIZE+i];
				unsigned int tmp = xorr - ((xorr >> 1) & 033333333333) - ((xorr >> 2) & 011111111111);
				count += ((tmp + (tmp >> 3)) & 030707070707) % 63;

				if(count > 1) break;
			}
			if (count == 1)
				goodPairsCount++;
		}
	}

	return goodPairsCount;
}

__global__ void
hammingKernel(int* sequenceTab, int* pairsTab, int size, unsigned int * stack, unsigned int * stackSize)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	if(tid >= size) return;
	int pair = pairsTab[tid];
	int x = pair >> 16;
	int y = pair & 0x0000FFFF;

	int count = 0;

	for(int i = 0; i < SIZE; i++)
	{
	    unsigned int xorr = sequenceTab[x*SIZE+i] ^ sequenceTab[y*SIZE+i];
		count += __popc(xorr);

	    if(count > 1) break;
	}

	if (count == 1)
	{
		int currIdx = atomicAdd(stackSize,2);
		stack[currIdx]=x;
		stack[currIdx+1]=y;
	}
}

__global__ void
hammingKernelNoPairsTab(int* sequenceTab, int numberOfSequences, unsigned int * stack, unsigned int * stackSize, int mask, int shift)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

	//int x = tid >> 16;
	//int y = tid & 0x0000FFFF;
	int x = (tid >> shift) & mask;
	int y = tid & mask;

	//if (x < y)
		//printf("Pair of ( %d , %d )\n", x, y);
	
	if (x >= numberOfSequences || y >= numberOfSequences)
		return;

	if (x >= y)
		return;

	int count = 0;
	for(int i = 0; i < SIZE; i++)
	{
	    unsigned int xorr = sequenceTab[x*SIZE+i] ^ sequenceTab[y*SIZE+i];
		count += __popc(xorr);

	    if(count > 1) break;
	}

	if (count == 1)
	{
		//printf("FOUND ONE\n");
		int currIdx = atomicAdd(stackSize,2);
		stack[currIdx]=x;
		stack[currIdx+1]=y;
	}
}

__global__ void
hammingKernelSelfPairs(int* sequenceTab, int n, unsigned int * stack, unsigned int * stackSize)
{
	int  threadRowId, threadColId;

   threadRowId = blockIdx.x * blockDim.x + threadIdx.x;
   threadColId = blockIdx.y * blockDim.y + threadIdx.y;

   if (threadRowId >= n || threadColId >= n /2)
	   return;

   if (threadColId >= threadRowId) {
	   threadRowId = n - 1 - threadRowId;
	   threadColId = n - 1 - threadColId;
   }
   
   //printf("Blk: (%d,%d) Thread: (%d,%d) -> Row/Col = (%d,%d)\n",  blockIdx.x, blockIdx.y,  threadIdx.x, threadIdx.y,  threadRowId, threadColId);

	int count = 0;
	for(int i = 0; i < SIZE; i++)
	{
	    unsigned int xorr = sequenceTab[threadColId*SIZE+i] ^ sequenceTab[threadRowId*SIZE+i];
		count += __popc(xorr);

	    if(count > 1) break;
	}

	if (count == 1)
	{
	    //printf(" Col/Row = (%d,%d)\n", threadColId, threadRowId);
		int currIdx = atomicAdd(stackSize,2);
		stack[currIdx]=threadColId;
		stack[currIdx+1]=threadRowId;
	}
}

////////////////////////////////////////////////////////////////////////////////
// Program main
int main(int argc, char **argv)
{
    runTest(argc, argv);
}

void
runTest(int argc, char **argv)
{
    srand(SEED);
    int n = NUMBER_OF_SEQUENCES;
    StopWatchInterface *timer = 0;
	sdkCreateTimer(&timer);

	//Variable for array of sequences
    int tabMemory = sizeof(int) * SIZE * n;
    int* hostSequenceTab = (int*)malloc(tabMemory);
    int* deviceSequenceTab;

	// Variables for Pairs and Results
	int pairsCount = n * (n-1) /2;
	printf("Pairs Count %d\n", pairsCount);
	int pairsMemory = sizeof(int) * pairsCount;
    int* hostPairsTab = (int*)malloc(pairsMemory);
    int* devicePairsTab;

	// Insert data to arrays
	FillSequenceTable(hostSequenceTab, n);

	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	FillPairTable(hostPairsTab, n);
	sdkStopTimer(&timer);
	printf("CREATING PAIR TABLE processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));

	// Copy memory to GPU
    checkCudaErrors(hipMalloc((void **) &deviceSequenceTab, tabMemory));
    checkCudaErrors(hipMemcpy(deviceSequenceTab, hostSequenceTab, tabMemory, hipMemcpyHostToDevice));
    checkCudaErrors(hipMalloc((void **) &devicePairsTab, pairsMemory));
    checkCudaErrors(hipMemcpy(devicePairsTab, hostPairsTab, pairsMemory, hipMemcpyHostToDevice));
    if(!deviceSequenceTab || !hostSequenceTab || !devicePairsTab || ! hostPairsTab)
		printf("Memory allocation failed\n");

	//Atomic Stack
	unsigned int * stackSize;
	hipMallocManaged(&stackSize, sizeof(unsigned int));
	*stackSize = 0;
	unsigned int * stack;
	hipMallocManaged(&stack,pairsMemory);
	if (!stack)
		printf("Stack memory alloc failed!\n");

	//METHODS

	//Calculations on GPU with PairsTab
    int tpb = THREAD_PER_BLOCK;
	int numberOfBlocks = pairsCount / tpb + 1;
	float sumOfGPUTimes = 0.f;

	for (int i = 0; i < TestCount; i++)
	{
		sdkResetTimer(&timer);
		sdkStartTimer(&timer);
		hammingKernel<<< numberOfBlocks, tpb>>>(deviceSequenceTab, devicePairsTab, pairsCount, stack, stackSize);
		hipDeviceSynchronize();
		sdkStopTimer(&timer);
		sumOfGPUTimes += sdkGetTimerValue(&timer);
	}

	//Print GPU results
	printf("METHOD : GETTING PAIR INDICES FROM PREVIOUSLY CREATED ARRAY\n");
	printf("[GPU] : Found [ %d ] pairs of Hamming Distance 1 \n", *stackSize / (2 * TestCount));
    printf("[GPU] Average GPU processing time for %d tests: %f (ms)\n\n", TestCount, sumOfGPUTimes / TestCount);

	//Calculations on GPU without PairsTab
	printf("METHOD : CALCULATING PAIR INDICES ON GPU\n");
	int max2 = MinimumBiggerPowerOf2(n);
	int mask = (int)pow(2, max2 + 1) - 1;
	int shift = max2 + 1;
	//PrintIntBinary(max2);
	int numOfThreads = (mask + 1) * (mask + 1);
	int numOfBlocks = numOfThreads / tpb + 1;
	printf("Shift is %d, mask is %d\n", shift, mask);
	float sumOfGPUTimesNoPairs = 0.f;
	for (int i = 0; i < TestCount; i++)
	{
		sdkResetTimer(&timer);
		sdkStartTimer(&timer);
		//hammingKernelNoPairsTab<<< 256 * 256 * 256, tpb>>>(deviceSequenceTab, n, stack, stackSize);
		hammingKernelNoPairsTab<<< numOfBlocks, tpb>>>(deviceSequenceTab, n, stack, stackSize, mask, shift);
		hipDeviceSynchronize();
		sdkStopTimer(&timer);
		sumOfGPUTimesNoPairs += sdkGetTimerValue(&timer);
	}
	
    printf("[GPU] Average GPU processing time with for %d tests: %f (ms)\n\n", TestCount, sumOfGPUTimesNoPairs / TestCount);

	printf("METHOD : CALCULATING PAIR INDICES ON GPU USING GRID OF BLOCKS\n");
	// Pair Finding implemented by creating grid of blocks
	int tpd = THREAD_PER_DIM;
	dim3 blockSize = dim3(tpd, tpd);
	int gridHeight = (n - 1) / tpd + 1;
	int gridWidth = gridHeight / 2 + 1;
	dim3 gridSize = dim3(gridHeight, gridWidth);
	sdkStartTimer(&timer);

	float sumOfGPUTimesSelfPairs = 0.f;
	for (int i = 0; i < TestCount; i++)
	{
		sdkResetTimer(&timer);
		sdkStartTimer(&timer);
		hammingKernelSelfPairs <<< gridSize, blockSize >>> (deviceSequenceTab, n, stack, stackSize);
		hipDeviceSynchronize();
		sdkStopTimer(&timer);
		sumOfGPUTimesSelfPairs += sdkGetTimerValue(&timer);
	}
    printf("[GPU] Average GPU processing time with for %d tests: %f (ms)\n\n", TestCount, sumOfGPUTimesSelfPairs / TestCount);

	//Calculations on CPU
	sdkResetTimer(&timer);
	sdkStartTimer(&timer);
	int goodPairsCountCPU = NumberOfGoodPairs(hostSequenceTab, n);
	sdkStopTimer(&timer);
	printf("METHOD : CALCULATIONS ON CPU\n");
	printf("[CPU] : Found [ %d ] pairs of Hamming Distance 1 \n", goodPairsCountCPU);
	printf("[CPU] processing time: %f (ms)\n", sdkGetTimerValue(&timer));

    PrintGoodPairsFromStack(hostSequenceTab, stack, *stackSize);
	//Free memory and timer
	sdkDeleteTimer(&timer);
    free(hostSequenceTab);
    free(hostPairsTab);
    hipFree(deviceSequenceTab);
    hipFree(devicePairsTab);
    getLastCudaError("Kernel execution failed");
}
